#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <fstream>
#include <chrono>
#include <limits.h>
#include <string>
#include <hip/hip_runtime.h>
#include "uthashgpu.h"
#define IN_PATH "F:\\Dev\\PLZW\\in.txt"
#define ALPHABET_LEN 256
#define DEFAULT_NBLOCKS 4
#define SHAREDMEM_MAX 64
#define MAX_TOKEN_SIZE 1000
#define CUDA_WARN(XXX) \
    do { if (XXX != hipSuccess) cerr << "CUDA Error: " << \
    hipGetErrorString(XXX) << ", at line " << __LINE__ \
    << endl; hipDeviceSynchronize(); } while (0)


using namespace std;

struct unsorted_node_map {
    char* id; // key
    unsigned int code;
    short tokenSize;
    UT_hash_handle hh; /* makes this structure hashable */
};

struct unsorted_node_map_dec {
    unsigned int id; // key
    char* token;
    short tokenSize;
    UT_hash_handle hh; /* makes this structure hashable */
};

__device__ unsorted_node_map* push_into_table(unsorted_node_map* table, char* id, short tokenSize, unsigned int code) {
    struct unsorted_node_map* s = (struct unsorted_node_map*)malloc(sizeof * s);
    char* curr_token = (char*)malloc((tokenSize + 1) * sizeof(char));
    memcpy(curr_token, id, tokenSize);
    curr_token[tokenSize] = '\0';
    s->id = curr_token;
    s->tokenSize = tokenSize;
    s->code = code;
    HASH_ADD_KEYPTR(hh, table, s->id, tokenSize, s);
    return table;
}

__device__ struct unsorted_node_map* find_by_token(unsorted_node_map* table, char* id, short length) {
    struct unsorted_node_map* s;
    id[length] = '\0';
    HASH_FIND_STR(table, id, s);
    return s;
}

__device__ struct unsorted_node_map* find_by_code(unsorted_node_map* table, unsigned int code) {
    struct unsorted_node_map* node, * tmp;
    HASH_ITER(hh, table, node, tmp) {
        if (node->code == code) {
            return node;
        }
    }
    return NULL;
}

__device__ void dispose_table(unsorted_node_map* table) {
    struct unsorted_node_map* node, * tmp;

    HASH_ITER(hh, table, node, tmp) {
        free(node->id);
        HASH_DEL(table, node);
        free(node);
    }
}

__device__ struct unsorted_node_map_dec* push_into_table_dec(unsorted_node_map_dec* table, unsigned int id, char* token, short tokenSize) {
    struct unsorted_node_map_dec* s = (struct unsorted_node_map_dec*)malloc(sizeof * s);
    char* curr_token = (char*)malloc((tokenSize + 1) * sizeof(char));
    memcpy(curr_token, token, tokenSize);
    curr_token[tokenSize] = '\0';
    s->token = curr_token;
    s->tokenSize = tokenSize;
    s->id = id;
    HASH_ADD_INT(table, id, s);
    return table;
}

__device__ struct unsorted_node_map_dec* find_by_code_dec(unsorted_node_map_dec* table, unsigned int id) {
    struct unsorted_node_map_dec* s;
    HASH_FIND_INT(table, &id, s);
    return s;
}

__device__ void dispose_table_dec(unsorted_node_map_dec* table) {
    struct unsorted_node_map_dec* node, * tmp;

    HASH_ITER(hh, table, node, tmp) {
        free(node->token);
        HASH_DEL(table, node);
        free(node);
    }
}

__device__ void loadEncodingCache(char* cache, unsigned int stripCacheLength, unsigned int cacheOffset, const char* globalItems, unsigned int globalItemsLength, unsigned int* nThreads, unsigned int thid) {
    unsigned int nitems = stripCacheLength * (cacheOffset + 1) <= globalItemsLength ? stripCacheLength : globalItemsLength - stripCacheLength * cacheOffset;
    //printf("th=%d --- nitems=%d\n", thid, nitems);
    for (unsigned int i = 0; i < nitems; i++) {
        cache[stripCacheLength * thid + i] = globalItems[cacheOffset * *nThreads * stripCacheLength + *nThreads * i + thid];
        //printf("th=%d --- cache[%d] = s1[%d]\n", thid, stripCacheLength * thid + i, cacheOffset * *nThreads * stripCacheLength + *nThreads * i + thid);
    }
    __syncthreads();
}

__device__ void loadDecodingCache(unsigned int* cache, unsigned int stripCacheLength, unsigned int cacheOffset, unsigned int* globalItems, unsigned int globalItemsLength, unsigned int thid) {
    unsigned int nitems = stripCacheLength * (cacheOffset + 1) <= globalItemsLength ? stripCacheLength : globalItemsLength - stripCacheLength * cacheOffset;
    //printf("th=%d --- nitems=%d\n", thid, nitems);
    for (unsigned int i = 0; i < nitems; i++) {
        cache[stripCacheLength * thid + i] = globalItems[cacheOffset * stripCacheLength + i ];
        //printf("th=%d --- cache[%d] = s1[%d]\n", thid, stripCacheLength * thid + i, cacheOffset * stripCacheLength + i);
    }
    __syncthreads();
}

__device__ int encoding_lzw(const char* s1, unsigned int count, unsigned int* objectCode, unsigned int avgRng, unsigned int* nThreads, unsigned int thid, char* cache, unsigned int stripCacheLength)
{
    struct unsorted_node_map* table;
    char* ch = (char*)malloc(sizeof(char));
    for (unsigned int i = 1; i < ALPHABET_LEN; i++) {
        ch[0] = char(i);
        // printf("tbl: %s %d\n", ch, i);
        table = push_into_table(table, ch, 1, i);
    }
    /*
    struct unsorted_node_map* noden1, * tmp1;
    HASH_ITER(hh, table, noden1, tmp1) {
        printf("tbl: %s %d\n", noden1->id, noden1->code);
    }*/
    free(ch);
    unsorted_node_map* node;
    int out_index = 0, pLength;
    char* p = (char*)malloc(MAX_TOKEN_SIZE * sizeof(char)), * pandc = (char*)malloc((MAX_TOKEN_SIZE + 1) * sizeof(char)), * c = new char[1];
    loadEncodingCache(cache, stripCacheLength, 0, s1, count, nThreads, thid);
    p[0] = cache[thid* stripCacheLength];
    p[1] = '\0';
    pandc[2] = '\0';
    pLength = 1;
    unsigned int code = ALPHABET_LEN, cacheOffset = 1, cacheIndex, nextCacheIndex;
    for (unsigned int i = 0; i < count; i++) {
        cacheIndex = i % stripCacheLength;
        nextCacheIndex = (i+1) % stripCacheLength;
        if (cacheIndex == stripCacheLength - 1) {
            loadEncodingCache(cache, stripCacheLength, cacheOffset, s1, count, nThreads, thid);
            cacheOffset++;
        }
        //printf("th=%d index=%d cacheIdx=%d cacheOffset=%d\n", thid, i, stripCacheLength * thid + cacheIndex, cacheOffset);
        if (i != count - 1) {
            c[0] = cache[stripCacheLength * thid + nextCacheIndex];
            //printf("th=%d i=%d accessing cache[%d] = %d\n", thid, i, stripCacheLength * thid + nextCacheIndex, cache[stripCacheLength * thid + nextCacheIndex]);
        }
        for (unsigned short str_i = 0; str_i < pLength; str_i++) pandc[str_i] = p[str_i];
        pandc[pLength] = c[0];
        unsorted_node_map* node = find_by_token(table, pandc, pLength + 1);
        //printf("%d %d, FINO QUI\n", thid, i);
        if (node != NULL) {
            p[++pLength] = '\0';
            pandc[pLength + 1] = '\0';
            for (unsigned short str_i = 0; str_i < pLength; str_i++) p[str_i] = pandc[str_i];
        }
        else {
            node = find_by_token(table, p, pLength);
            objectCode[out_index++] = node->code;
            table = push_into_table(table, pandc, pLength + 1, code);
            code++;
            p[0] = c[0];
            if (pLength > 1) {
                p[1] = '\0';
                pandc[2] = '\0';
            }
            pLength = 1;
        }
        c[0] = NULL;
    }
    objectCode[out_index++] = find_by_token(table, p, pLength)->code;
    /*
    struct unsorted_node_map* noden, * tmp;

    HASH_ITER(hh, table, noden, tmp) {
        if(noden->code > 255)
            printf("tbl: %s %d\n", noden->id, noden->code);
    }*/
    free(p);
    free(pandc);
    dispose_table(table);
    return out_index;
}
__device__ unsigned int decoding_lzw(unsigned int* op, char* decodedData, unsigned int* encodedBuffLengths, unsigned int* nThreads, unsigned int thid, unsigned int* cache, unsigned int stripCacheLength)
{
    struct unsorted_node_map_dec* table;
    char* ch = (char*)malloc(sizeof(char));
    for (unsigned int i = 1; i < ALPHABET_LEN; i++) {
        ch[0] = char(i);
        table = push_into_table_dec(table, i, ch, 1);
    }
    free(ch);

    unsigned int old, decodedDataLength, n, cacheIndex, nextCacheIndex, cacheOffset;
    loadDecodingCache(cache, stripCacheLength, 0, op, encodedBuffLengths[thid], thid);
    old = cache[stripCacheLength * thid];
    cacheOffset = 1;
    struct unsorted_node_map_dec* temp_node, * s_node = find_by_code_dec(table, old);
    int temp_length = 0, s_length = s_node->tokenSize;
    char* s = (char*)malloc(MAX_TOKEN_SIZE * sizeof(char)), * temp = (char*)malloc(sizeof(char));
    memcpy(s, s_node->token, s_length);
    s[s_length] = '\0';
    temp[0] = '\0';
    char c = s[0];

    memcpy(decodedData, s, s_length);
    decodedDataLength = 1;
    int count = ALPHABET_LEN;
    for (int i = 0; i < encodedBuffLengths[thid] - 1; i++) {
        cacheIndex = i % stripCacheLength;
        nextCacheIndex = (i + 1) % stripCacheLength;
        if (cacheIndex == stripCacheLength - 1) {
            loadDecodingCache(cache, stripCacheLength, cacheOffset, op, encodedBuffLengths[thid], thid);
            cacheOffset++;
        }
        n = cache[stripCacheLength * thid + nextCacheIndex];
        //printf("th=%d i=%d accessing cache[%d] = %d\n", thid, i, stripCacheLength * thid + nextCacheIndex, cache[stripCacheLength * thid + nextCacheIndex]);
        if (find_by_code_dec(table, n) == NULL) {
            s_node = find_by_code_dec(table, old);
            s_length = s_node->tokenSize;
            s_length++;
            memcpy(s, s_node->token, s_length - 1);
            s[s_length - 1] = c;
            s[s_length] = '\0';
        }
        else {
            s_node = find_by_code_dec(table, n);
            if (s_node->tokenSize != s_length) {
                s_length = s_node->tokenSize;
                s[s_length] = '\0';
            }
            memcpy(s, s_node->token, s_length);
        }
        memcpy(&decodedData[decodedDataLength], s, s_length);
        decodedDataLength += s_length;
        c = s[0];
        temp_node = find_by_code_dec(table, old);
        if (temp_length != temp_node->tokenSize + 1) {
            temp_length = temp_node->tokenSize + 1;
            temp[temp_length] = '\0';
        }
        memcpy(temp, temp_node->token, temp_length - 1);
        temp[temp_length - 1] = c;
        table = push_into_table_dec(table, count, temp, temp_length);
        count++;
        old = n;
    }
    free(temp);
    free(s);
    dispose_table_dec(table);
    return decodedDataLength;
}

__global__ void encoding(char *input, unsigned int *inputLength, unsigned int *encodedData, unsigned int* encodedBuffLengths, unsigned int* nThreads) {
    unsigned int sharedItems_MAX, thid = threadIdx.x + blockIdx.x * blockDim.x;
    sharedItems_MAX = SHAREDMEM_MAX; //*sharedMem_MAX / (sizeof(char));
    extern __shared__ char enc_cache[SHAREDMEM_MAX];
    sharedItems_MAX /= *nThreads;

    unsigned int encodedLength, dataBuffLength, * encodedBuff,
        avgRng = __double2uint_ru((double)(*inputLength) / (double)(*nThreads)), avgRngRest = *inputLength % *nThreads;
    dataBuffLength = avgRngRest == 0 || thid < avgRngRest ? avgRng : avgRng - 1;

    unsigned int encOffset = 0,* encodedDataBuff = new unsigned int[dataBuffLength];

    encodedBuffLengths[thid] = encoding_lzw(input, dataBuffLength, encodedDataBuff, avgRng, nThreads, thid, enc_cache, sharedItems_MAX);
    __syncthreads();

    for (unsigned int i = 0; i < thid; i++) {
        encOffset += encodedBuffLengths[i];
    }
    for (unsigned int i = 0; i < encodedBuffLengths[thid]; i++) {
        encodedData[encOffset + i] = encodedDataBuff[i];
        //printf("th%d i=%d  %d\n", thid, i, encodedDataBuff[i]);
    }

}

__global__ void decoding(unsigned int* encodedData, unsigned int* encodedBuffLengths, unsigned int* inputLength, char* decodedData, unsigned int* decodedBuffLengths, unsigned int* nThreads) {
    unsigned int sharedItems_MAX, thid = threadIdx.x + blockIdx.x * blockDim.x;
    sharedItems_MAX = SHAREDMEM_MAX; //*sharedMem_MAX / (sizeof(char));

    extern __shared__ unsigned int dec_cache[SHAREDMEM_MAX];
    sharedItems_MAX /= *nThreads;

    unsigned int encodedLength = 0, dataBuffLength, *encodedBuff, encodedOffset = 0,
        avgRng = __double2uint_ru((double)(*inputLength) / (double)(*nThreads)), avgRngRest = *inputLength % *nThreads;
    dataBuffLength = avgRngRest == 0 || thid < avgRngRest ? avgRng : avgRng - 1;

    char* decodedDataBuff = new char[dataBuffLength];

    for (unsigned int i = 0; i < thid; i++) {
        encodedOffset += encodedBuffLengths[i];
    }

    decodedBuffLengths[thid] = decoding_lzw(&encodedData[encodedOffset], decodedDataBuff, encodedBuffLengths, nThreads, thid, dec_cache, sharedItems_MAX);
    __syncthreads();

    for (unsigned int i = 0; i < dataBuffLength; i++) {
        decodedData[*nThreads * i + thid] = decodedDataBuff[i];
        //printf("th%d i=%d  %d\n", thid, i, decodedDataBuff[i]);
    }

    //delete[] decodedDataBuff;
}

int main()
{
    hipDeviceProp_t prop;
    int count, sharedMem_MAX, nBlocks_MAX, warpSize;
    hipGetDeviceCount(&count);
    if (count > 0) {
        hipGetDeviceProperties(&prop, 0); // getting first device props
        sharedMem_MAX = prop.sharedMemPerBlock; // 49152 bytes per block for GTX 1070 (capability 6.1)
        nBlocks_MAX = prop.maxGridSize[0]; // 2147483647 blocks for GTX 1070 (capability 6.1)
        warpSize = 3; //prop.warpSize;
    }
    else {
        cout << "No device detected" << endl;
        exit(1);
    }

    string input;
    string line;
    ifstream inFile;
    bool correctness = true;
    inFile.open(IN_PATH);
    if (!inFile) {
        cout << "Unable to open file";
        exit(1);
    }
    while (inFile >> line) {
        input += line;
    }
    inFile.close();

    unsigned int nBlocks, nThreads, inputLength = input.length(), inputSize = inputLength * sizeof(char),
        *dev_encodedData, *dev_inputLength, *dev_encodedBuffLengths, *dev_nThreads, *dev_sharedMem_MAX,
        *encodedData = (unsigned int*)malloc(inputLength * sizeof(unsigned int)), encodedLength = 0,
        *encodedBuffLengths;
    char* dev_input;
    const char* input_point = input.c_str();

    std::chrono::steady_clock::time_point encoding_begin, encoding_end, decoding_begin, decoding_end;
    encoding_begin = std::chrono::steady_clock::now();
    nBlocks = 1; // DEFAULT_NBLOCKS;
    nThreads = nBlocks * warpSize;
    encodedBuffLengths = (unsigned int*)malloc(nThreads * sizeof(unsigned int));
    
    CUDA_WARN(hipMalloc((void**)&dev_input, inputLength * sizeof(char)));
    CUDA_WARN(hipMalloc((void**)&dev_inputLength, sizeof(unsigned int)));
    CUDA_WARN(hipMalloc((void**)&dev_nThreads, sizeof(unsigned int)));
    CUDA_WARN(hipMalloc((void**)&dev_encodedData, inputLength * sizeof(unsigned int)));
    CUDA_WARN(hipMalloc((void**)&dev_encodedBuffLengths, nThreads * sizeof(unsigned int)));

    CUDA_WARN(hipMemcpy(dev_input, input_point, inputLength * sizeof(char), hipMemcpyHostToDevice));
    CUDA_WARN(hipMemcpy(dev_inputLength, &inputLength, sizeof(unsigned int), hipMemcpyHostToDevice));
    CUDA_WARN(hipMemcpy(dev_nThreads, &nThreads, sizeof(unsigned int), hipMemcpyHostToDevice));

    encoding<<< nBlocks, warpSize >>>(dev_input, dev_inputLength, dev_encodedData, dev_encodedBuffLengths, dev_nThreads);

    CUDA_WARN(hipMemcpy(encodedData, dev_encodedData, inputLength * sizeof(unsigned int), hipMemcpyDeviceToHost));
    CUDA_WARN(hipMemcpy(encodedBuffLengths, dev_encodedBuffLengths, nThreads * sizeof(unsigned int), hipMemcpyDeviceToHost));

    for (unsigned int i = 0; i < nThreads; i++) {
        encodedLength += encodedBuffLengths[i];
    }

    CUDA_WARN(hipFree(dev_input));
    CUDA_WARN(hipFree(dev_inputLength));
    CUDA_WARN(hipFree(dev_encodedData));
    CUDA_WARN(hipFree(dev_encodedBuffLengths));
    CUDA_WARN(hipFree(dev_nThreads));
    
    encoding_end = std::chrono::steady_clock::now();

    //for (unsigned int i = 0; i < encodedLength; i++) printf("%d ", encodedData[i]);
    printf("\n\n");
    char* decodedData = (char*)malloc(inputSize);
    unsigned int* decodedBuffLengths = (unsigned int*)malloc(nThreads * sizeof(unsigned int));
    decoding_begin = std::chrono::steady_clock::now();
    char* dev_decodedData;
    unsigned int *dev_decodedBuffLengths, decodedDataLength = 0;

    CUDA_WARN(hipMalloc((void**)&dev_encodedData, encodedLength * sizeof(unsigned int)));
    CUDA_WARN(hipMalloc((void**)&dev_encodedBuffLengths, nThreads * sizeof(unsigned int)));
    CUDA_WARN(hipMalloc((void**)&dev_inputLength, sizeof(unsigned int)));
    CUDA_WARN(hipMalloc((void**)&dev_nThreads, sizeof(unsigned int)));
    CUDA_WARN(hipMalloc((void**)&dev_decodedData, inputSize));
    CUDA_WARN(hipMalloc((void**)&dev_decodedBuffLengths, nThreads * sizeof(unsigned int)));

    CUDA_WARN(hipMemcpy(dev_encodedData, encodedData, encodedLength * sizeof(unsigned int), hipMemcpyHostToDevice));
    CUDA_WARN(hipMemcpy(dev_encodedBuffLengths, encodedBuffLengths, nThreads * sizeof(unsigned int), hipMemcpyHostToDevice));
    CUDA_WARN(hipMemcpy(dev_inputLength, &inputLength, sizeof(unsigned int), hipMemcpyHostToDevice));
    CUDA_WARN(hipMemcpy(dev_nThreads, &nThreads, sizeof(unsigned int), hipMemcpyHostToDevice));

    decoding<<< nBlocks, warpSize >>>(dev_encodedData, dev_encodedBuffLengths, dev_inputLength, dev_decodedData, dev_decodedBuffLengths, dev_nThreads);

    CUDA_WARN(hipMemcpy(decodedData, dev_decodedData, inputSize, hipMemcpyDeviceToHost));
    CUDA_WARN(hipMemcpy(decodedBuffLengths, dev_decodedBuffLengths, nThreads * sizeof(unsigned int), hipMemcpyDeviceToHost));

    for (unsigned int i = 0; i < nThreads; i++) {
        decodedDataLength += decodedBuffLengths[i];
    }

    CUDA_WARN(hipFree(dev_encodedData));
    CUDA_WARN(hipFree(dev_encodedBuffLengths));
    CUDA_WARN(hipFree(dev_inputLength));
    CUDA_WARN(hipFree(dev_nThreads));
    CUDA_WARN(hipFree(dev_decodedData));
    CUDA_WARN(hipFree(dev_decodedBuffLengths));

    decoding_end = std::chrono::steady_clock::now();

    if (inputLength == decodedDataLength) {
        for (unsigned int j = 0; j < inputLength; j++) {
            correctness = input[j] == decodedData[j];
            if (correctness == 0) {
                break;
            }
        }
    }
    else {
        correctness = 0;
    }

    cout << "Lossless propriety: " << correctness;

    cout <<
        "\nChars: " << inputLength << "  Memory: " << inputLength * sizeof(char) << " bytes" <<
        "\nEncoded: " << encodedLength << "  Memory: " << encodedLength * sizeof(unsigned int) << " bytes" << endl;


    cout << "Encoding time: " << std::chrono::duration_cast<std::chrono::milliseconds> (encoding_end - encoding_begin).count() << "[ms]" << std::endl;
    cout << "Decoding time: " << std::chrono::duration_cast<std::chrono::milliseconds> (decoding_end - decoding_begin).count() << "[ms]" << std::endl;

    free(encodedData);
    free(decodedData);
    return 0;
}
