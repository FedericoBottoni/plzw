#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <fstream>
#include <chrono>
#include <limits.h>
#include <string>
#include <hip/hip_runtime.h>
#include "../dependencies/uthash.h"
#define IN_PATH "F:\\Dev\\PLZW\\in.txt"
#define ALPHABET_LEN 256
#define DEFAULT_NBLOCKS 4
#define MAX_TOKEN_SIZE 1000

using namespace std;

struct unsorted_node_map {
    char *id; // key
    unsigned int code;
    short tokenSize;
    UT_hash_handle hh; /* makes this structure hashable */
};

struct unsorted_node_map* table = NULL;

void push_into_table(char* id, short tokenSize, unsigned int code) {
    struct unsorted_node_map* s = (struct unsorted_node_map*)malloc(sizeof * s);
    char* curr_token = (char*)malloc(tokenSize * sizeof(char));
    memcpy(curr_token, id, tokenSize);
    s->id = curr_token;
    s->tokenSize = tokenSize;
    s->code = code;
    HASH_ADD_KEYPTR(hh, table, s->id, tokenSize, s);
}

struct unsorted_node_map* find_by_token(char* id, short length) {
    struct unsorted_node_map* s = (struct unsorted_node_map*)malloc(sizeof * s);
    HASH_FIND_STR(table, id, s);
    return s;
}

struct unsorted_node_map* find_by_code(unsigned int code) {
    struct unsorted_node_map* s = (struct unsorted_node_map*)malloc(sizeof * s);
    HASH_FIND_INT(table, &code, s);
    return s;
}

void dispose_table() {
    struct unsorted_node_map* node = (struct unsorted_node_map*)malloc(sizeof * node),
        *tmp = (struct unsorted_node_map*)malloc(sizeof * tmp);

    HASH_ITER(hh, table, node, tmp) {
        free(node->id);
        HASH_DEL(table, node);
        free(node);
    }
}


int encoding_lzw(const char* s1, unsigned int count, unsigned int* objectCode)
{
    char* ch = (char*)malloc(sizeof(char));
    for (unsigned int i = 1; i < ALPHABET_LEN; i++) {
        ch[0] = char(i);
        push_into_table(ch, 1, i);
    }

    int out_index = 0, pLength;
    char* p = (char*)malloc(sizeof(char)), * pandc = (char*)malloc(2*sizeof(char)), *c = new char[1];
    memset(p, 0, sizeof(p));
    memset(pandc, 0, sizeof(p));
    p[0] = s1[0];
    pLength = 1;
    unsigned int code = ALPHABET_LEN;
    unsigned int i;
    for (i = 0; i < count; i++) {
        if (i != count - 1)
            c[0] = s1[i + 1];
        for (unsigned short str_i = 0; str_i < pLength; str_i++) pandc[str_i] = p[str_i];
        pandc[pLength] = c[0];
        if (find_by_token(pandc, pLength + 1) != NULL) {
            p = (char*)realloc(p, ++pLength * sizeof(char));
            pandc = (char*)realloc(pandc, (pLength+1) * sizeof(char));
            for (unsigned short str_i = 0; str_i < pLength; str_i++) p[str_i] = pandc[str_i];
        }
        else {
            unsorted_node_map *node = find_by_token(p, pLength);
            objectCode[out_index++] = node->code;
            push_into_table(pandc, pLength + 1, code);
            code++;
            p[0] = c[0];
            if (pLength > 1) {
                p = (char*)realloc(p, sizeof(char));
                pandc = (char*)realloc(pandc, 2*sizeof(char));
            }
            pLength = 1;
        }
        c[0] = NULL;
        //memset(pandc, 0, sizeof(pandc));
    }
    objectCode[out_index] = find_by_token(p, pLength)->code;

    free(p);
    free(pandc);
    dispose_table();
    return out_index;
}

unsigned int decoding_lzw(unsigned int* op, int op_length, char* decodedData)
{
    char ch;
    for (unsigned int i = 0; i < ALPHABET_LEN; i++) {
        ch = char(i);
        push_into_table(&ch, 1, i);
    }

    unsigned int old = op[0], decodedDataLength, n;
    struct unsorted_node_map* temp_node, * s_node = find_by_code(old);
    int temp_length, s_length = s_node->tokenSize;
    char* s = new char[MAX_TOKEN_SIZE], * temp = new char[MAX_TOKEN_SIZE];
    memcpy(s, s_node->id, s_length);
    char* c = s;
    memcpy(decodedData, s, s_length);
    decodedDataLength = 1;
    int count = ALPHABET_LEN;
    for (int i = 0; i < op_length - 1; i++) {
        n = op[i + 1];
        if (find_by_token(s, s_length) == NULL) {
            s_node = find_by_code(old);
            s_length = s_node->tokenSize;
            memcpy(s, s_node->id, s_length);
            s[s_length++] = *c;
        }
        else {
            s_node = find_by_code(n);
            s_length = s_node->tokenSize;
            memcpy(s, s_node->id, s_length);
        }
        if (s_length > MAX_TOKEN_SIZE - 1) {
            printf("Token-size is not enough big");
            exit(-1);
        }
        memcpy(&decodedData[decodedDataLength], s, s_length);
        decodedDataLength += s_length;
        c = s;
        temp_node = find_by_code(old);
        temp_length = temp_node->tokenSize;
        memcpy(temp, temp_node->id, temp_length);
        temp[temp_length] = *c;
        push_into_table(temp, temp_length + 1, count);
        count++;
        old = n;
    }
    delete[] temp;
    delete[] s;
    dispose_table();
    return decodedDataLength;
}



__global__ void encoding(char *input, unsigned int *inputLength, unsigned int *encodedData, unsigned int* nBlocks) {
    unsigned int block = blockIdx.x;
    char thid = threadIdx.x;

    //extern __shared__ unsigned int *cache_shared[];
    //unsigned int* cacheStart = cache_shared[0], *cacheEnd = cache_shared[1], *cache = cache_shared[2];

    //printf("tid = %d\n", thid);
}


int main()
{
    hipDeviceProp_t prop;
    int count, sharedMem_MAX, nBlocks_MAX;
    hipGetDeviceCount(&count);
    if (count > 0) {
        hipGetDeviceProperties(&prop, 0); // getting first device props
        sharedMem_MAX = prop.sharedMemPerBlock; // 49152 bytes per block for GTX 1070 (capability 6.1)
        nBlocks_MAX = prop.maxGridSize[0]; // 2147483647 blocks for GTX 1070 (capability 6.1)
    }
    else {
        cout << "No device detected" << endl;
        exit(1);
    }

    string input;
    string line;
    ifstream inFile;
    bool correctness = true;
    inFile.open(IN_PATH);
    if (!inFile) {
        cout << "Unable to open file";
        exit(1);
    }
    while (inFile >> line) {
        input += line;
    }
    inFile.close();

    unsigned int nBlocks, inputLength = input.length(), inputSize = inputLength * sizeof(char);
    unsigned int *dev_encodedData, *dev_inputLength, *dev_nBlocks;
    unsigned int* encodedData = (unsigned int*)malloc(inputLength * sizeof(unsigned int));
    char* dev_input;
    std::chrono::steady_clock::time_point encoding_begin, encoding_end, decoding_begin, decoding_end;

    encoding_begin = std::chrono::steady_clock::now();
    const char *input_point = input.c_str();
    nBlocks = DEFAULT_NBLOCKS;
    /*
    hipMalloc((void**)&dev_input, inputLength * sizeof(char));
    hipMalloc((void**)&dev_inputLength, sizeof(unsigned int));
    hipMalloc((void**)&dev_encodedData, inputLength * sizeof(char));
    hipMalloc((void**)&dev_nBlocks, sizeof(unsigned int));
    hipMemcpy(dev_input, input_point, inputLength * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(dev_inputLength, &inputLength, sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(dev_nBlocks, &nBlocks, sizeof(unsigned int), hipMemcpyHostToDevice);

    encoding<<< 3, nBlocks >>>(dev_input, dev_inputLength, dev_encodedData, dev_nBlocks);

    hipMemcpy(encodedData, dev_input, inputLength * sizeof(unsigned int), hipMemcpyDeviceToHost);

    hipFree(dev_input);
    hipFree(dev_inputLength);
    hipFree(dev_encodedData);
    hipFree(dev_nBlocks);
    */
    unsigned int encodedLength = encoding_lzw(input_point, input.length(), encodedData);
    encoding_end = std::chrono::steady_clock::now();

    encodedData = (unsigned int*)realloc(encodedData, (encodedLength) * sizeof(unsigned int));
    char* decodedData = (char*)malloc(inputSize);
    //for (unsigned int j = 0; j < encodedLength; j++) {
    //    cout << encodedData[j] << " ";
    //}

    decoding_begin = std::chrono::steady_clock::now();
    unsigned int decodedDataLength = decoding_lzw(encodedData, encodedLength, decodedData);
    decoding_end = std::chrono::steady_clock::now();

    // cout << decodedData << "\n\n";

    if (inputLength == decodedDataLength) {
        for (unsigned int j = 0; j < inputLength; j++) {
            correctness = input[j] == decodedData[j];
            if (correctness == 0) {
                break;
            }
        }
    }
    else {
        correctness = 0;
    }

    cout << "Lossless propriety: " << correctness;

    cout <<
        "\nChars: " << inputLength << "  Memory: " << inputLength * sizeof(char) << " bytes" <<
        "\nEncoded: " << encodedLength << "  Memory: " << encodedLength * sizeof(unsigned int) << " bytes" << endl;


    cout << "Encoding time: " << std::chrono::duration_cast<std::chrono::milliseconds> (encoding_end - encoding_begin).count() << "[ms]" << std::endl;
    cout << "Decoding time: " << std::chrono::duration_cast<std::chrono::milliseconds> (decoding_end - decoding_begin).count() << "[ms]" << std::endl;

    delete[] encodedData;
    return 0;
}
