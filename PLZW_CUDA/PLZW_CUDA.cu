#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <fstream>
#include <chrono>
#include <limits.h>
#include <unordered_map>
#include <string>
#include <hip/hip_runtime.h>
#define IN_PATH "F:\\Dev\\PLZW\\in.txt"
#define ALPHABET_LEN 256
#define DEFAULT_NBLOCKS 4
#define MAX_TOKEN_SIZE 1000

struct unordered_map_node
{
    char* token;
    unsigned int code;
    struct unordered_map_node* next;
};

struct unordered_map_node* unordered_map_head = NULL;

void unordered_map_push(char* token, int code, unsigned int tokenLength) {
    struct unordered_map_node* link = (struct unordered_map_node*)malloc(sizeof(struct unordered_map_node));
    char* token_pointer = (char*)malloc(tokenLength * sizeof(char));
    memcpy(token_pointer, token, tokenLength);
    link->token = token_pointer;
    link->code = code;
    link->next = unordered_map_head;
    //printf("a %s\n", link);
    unordered_map_head = link;
    //printf("b %s\n", unordered_map_head);
}

unsigned int getCodeFromMap(char* token, int tokenLength) {
    struct unordered_map_node* ptr = unordered_map_head;

    while (ptr != NULL) {
        bool equals = true;
        for (unsigned int i = 0; i < tokenLength; i++) {
            char currentTokenChar = ptr->token[i];
            if (currentTokenChar != token[i]) {
                equals = false;
                break;
            }
        }
        if (equals) {
            return ptr->code;
        }
        else {
            ptr = ptr->next;
        }
    }
    return UINT_MAX;
}

char* getTokenFromMap(unsigned int code) {
    struct unordered_map_node* ptr = unordered_map_head;

    while (ptr != NULL) {
        bool equals = true;
        if (ptr->code == code) {
            return ptr->token;
        }
        else {
            ptr = ptr->next;
        }
    }
    return NULL;
}

bool isTokenInMap(char* token, int tokenLength) {
    unsigned int code = getCodeFromMap(token, tokenLength);
    return code != UINT_MAX;
}

bool isCodeInMap(unsigned int code) {
    char* token = getTokenFromMap(code);
    return token != NULL;
}

int encoding_lzw(const char* s1, unsigned int count, unsigned int* objectCode)
{
    int mapLength = ALPHABET_LEN;
    char* ch;
    for (unsigned int i = 0; i < ALPHABET_LEN; i++) {
        ch = new char[1];
        ch[0] = char(i);
        unordered_map_push(ch, i, 1);
    }
    delete[] ch;

    int out_index = 0, pLength;
    char *p = new char[MAX_TOKEN_SIZE], * pandc = new char[MAX_TOKEN_SIZE], *c = new char[1];
    p[0] = s1[0];
    pLength = 1;
    unsigned int code = ALPHABET_LEN;
    unsigned int i;
    for (i = 0; i < count; i++) {
        if (i != count - 1)
            c[0] = s1[i + 1];
        pandc = strncpy(pandc, p, pLength);
        pandc[pLength] = c[0];
        if (isTokenInMap(pandc, pLength + 1)) {
            strcpy(p, pandc);
            pLength++;
        }
        else {
            objectCode[out_index++] = getCodeFromMap(p, pLength);
            unordered_map_push(pandc, code, pLength + 1);
            code++;
            memset(p, 0, sizeof(p));
            p[0] = c[0];
            pLength = 1;
        }
        memset(c, 0, sizeof(c));
        memset(pandc, 0, sizeof(pandc));
    }
    objectCode[out_index] = getCodeFromMap(p, pLength);
    return out_index;
}


__global__ void encoding(char *input, unsigned int *inputLength, unsigned int *encodedData, unsigned int* nBlocks) {
    unsigned int block = blockIdx.x;
    char thid = threadIdx.x;

    //extern __shared__ unsigned int *cache_shared[];
    //unsigned int* cacheStart = cache_shared[0], *cacheEnd = cache_shared[1], *cache = cache_shared[2];

    //printf("tid = %d\n", thid);
}

using namespace std;
int main()
{
    hipDeviceProp_t prop;
    int count, sharedMem_MAX, nBlocks_MAX;
    hipGetDeviceCount(&count);
    if (count > 0) {
        hipGetDeviceProperties(&prop, 0); // getting first device props
        sharedMem_MAX = prop.sharedMemPerBlock; // 49152 bytes per block for GTX 1070 (capability 6.1)
        nBlocks_MAX = prop.maxGridSize[0]; // 2147483647 blocks for GTX 1070 (capability 6.1)
    }
    else {
        cout << "No device detected" << endl;
        exit(1);
    }

    string input;
    string line;
    ifstream inFile;
    bool correctness = true;
    inFile.open(IN_PATH);
    if (!inFile) {
        cout << "Unable to open file";
        exit(1);
    }
    while (inFile >> line) {
        input += line;
    }
    inFile.close();

    unsigned int nBlocks, inputLength = input.length();
    unsigned int *dev_encodedData, *dev_inputLength, *dev_nBlocks, *encodedData = new unsigned int[inputLength];
    char* dev_input;
    std::chrono::steady_clock::time_point encoding_begin, encoding_end, decoding_begin, decoding_end;

    encoding_begin = std::chrono::steady_clock::now();
    const char *input_point = input.c_str();
    nBlocks = DEFAULT_NBLOCKS;

    hipMalloc((void**)&dev_input, inputLength * sizeof(char));
    hipMalloc((void**)&dev_inputLength, sizeof(unsigned int));
    hipMalloc((void**)&dev_encodedData, inputLength * sizeof(char));
    hipMalloc((void**)&dev_nBlocks, sizeof(unsigned int));
    hipMemcpy(dev_input, input_point, inputLength * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(dev_inputLength, &inputLength, sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(dev_nBlocks, &nBlocks, sizeof(unsigned int), hipMemcpyHostToDevice);

    encoding<<< 3, nBlocks >>>(dev_input, dev_inputLength, dev_encodedData, dev_nBlocks);

    hipMemcpy(encodedData, dev_input, inputLength * sizeof(unsigned int), hipMemcpyDeviceToHost);

    hipFree(dev_input);
    hipFree(dev_inputLength);
    hipFree(dev_encodedData);
    hipFree(dev_nBlocks);

    unsigned int encodedLength = encoding_lzw(input_point, input.length(), encodedData);
    encoding_end = std::chrono::steady_clock::now();

    encodedData = (unsigned int*)realloc(encodedData, (encodedLength) * sizeof(unsigned int));
    //for (unsigned int j = 0; j < encodedLength; j++) {
    //    cout << encodedData[j] << " ";
    //}

    decoding_begin = std::chrono::steady_clock::now();
    string decodedData = ""; //decoding_lzw(encodedData, encodedLength);
    decoding_end = std::chrono::steady_clock::now();

    // cout << decodedData << "\n\n";

    if (inputLength == decodedData.length()) {
        for (unsigned int j = 0; j < inputLength; j++) {
            correctness = input[j] == decodedData[j];
            if (correctness == 0) {
                break;
            }
        }
    }
    else {
        correctness = 0;
    }

    cout << "Lossless propriety: " << correctness;

    cout <<
        "\nChars: " << inputLength << "  Memory: " << inputLength * sizeof(char) << " bytes" <<
        "\nEncoded: " << encodedLength << "  Memory: " << encodedLength * sizeof(unsigned int) << " bytes" << endl;


    cout << "Encoding time: " << std::chrono::duration_cast<std::chrono::milliseconds> (encoding_end - encoding_begin).count() << "[ms]" << std::endl;
    cout << "Decoding time: " << std::chrono::duration_cast<std::chrono::milliseconds> (decoding_end - decoding_begin).count() << "[ms]" << std::endl;

    delete[] encodedData;
    return 0;
}
